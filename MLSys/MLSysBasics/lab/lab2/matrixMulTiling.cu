#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_WIDTH 16  // Tile width for tiling

// Kernel for matrix multiplication using tiling
__global__ void matrixMulTiled(float *A, float *B, float *C, int M, int N, int P) {
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float value = 0.0f;

    for (int t = 0; t < (N - 1) / TILE_WIDTH + 1; ++t) {
        if (row < M && (t * TILE_WIDTH + threadIdx.x) < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_WIDTH + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < P && (t * TILE_WIDTH + threadIdx.y) < N)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * P + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i)
            value += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];

        __syncthreads();
    }

    if (row < M && col < P)
        C[row * P + col] = value;
}

// Function to print a matrix
void printMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

// Matrix multiplication with performance monitoring and printing input/output
void matrixMultiplication(float *h_A, float *h_B, float *h_C, int M, int N, int P) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * N * sizeof(float);
    size_t size_B = N * P * sizeof(float);
    size_t size_C = M * P * sizeof(float);

    // Allocate device memory
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Set grid and block dimensions
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((P - 1) / TILE_WIDTH + 1, (M - 1) / TILE_WIDTH + 1, 1);

    // CUDA event variables for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    // Launch the kernel
    matrixMulTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, P);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix multiplication (tiled) execution time: %f ms\n", milliseconds);

    // Copy the result from device to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    // Example matrix sizes
    int M = 4;  // Number of rows in A and C
    int N = 4;  // Number of columns in A and rows in B
    int P = 4;  // Number of columns in B and C

    // Allocate host memory
    float *h_A = (float*)malloc(M * N * sizeof(float));
    float *h_B = (float*)malloc(N * P * sizeof(float));
    float *h_C = (float*)malloc(M * P * sizeof(float));

    // Initialize matrices A and B with some values
    for (int i = 0; i < M * N; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < N * P; i++) {
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Print the input matrices
    printf("Matrix A:\n");
    printMatrix(h_A, M, N);

    printf("Matrix B:\n");
    printMatrix(h_B, N, P);

    // Perform matrix multiplication
    matrixMultiplication(h_A, h_B, h_C, M, N, P);

    // Print the output matrix
    printf("Matrix C (Result):\n");
    printMatrix(h_C, M, P);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
